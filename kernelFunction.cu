#include "hip/hip_runtime.h"
﻿#include "CUDA_CAU_HEARER.h"

typedef unsigned int UINT;

__global__ void myCudaVectorAdd()
{

    return ;
}

__global__ void myCudaVectorMultiple()
{

    return;
}

__global__ void myCudaMatrixAdd()
{

    return;
}

__global__ void myCudaMatrixMultiple(
	float* data1,
	float* data2,
	size_t row1,
	size_t col1,
	size_t row2,
	size_t col2,
	float* ret
)
{
	UINT currRow = blockIdx.x * blockDim.x + threadIdx.x;
	UINT currCol = blockIdx.y * blockDim.y + threadIdx.y;
	float sum = 0.0;
	for (size_t j = 0; j < col1; j++)
	{
		sum += data1[currRow * col1 + j] * data2[j * col2 + currCol];
	}
	ret[currRow * col2 + currCol] = sum;	
	return;
}
