#include "baseHeader.h"

OUT_HOST_PTR PDATA_SET makeDataSet(
	IN myCudaDataFlag inputDataFlag,
	IN DATA_GENERATE_TYPE dataGenerateType,
	IN SIZE_T (*dimensions)[MAX_DIMENSION_SIZE],
	IN_OPT PVOID existData
)
{
	if ((*dimensions) == NULL)
	{
		printf("���Ϸ���ά�����룬�Ѳ���.\n");
		exit(0xA000);
	}
	PDATA_SET ret = (PDATA_SET)malloc(sizeof(DATA_SET));
	ret->dataFlag = inputDataFlag;
	if (ret->dataFlag == CUDA_DATA_FLAG_$_VECTOR_$)
	{
		if ((*dimensions)[1] != 0 || (*dimensions)[2] != 0)
		{
			printf("����ά�����������Ϳ��ܲ������Ѳ���.\n");
			free(ret);
			ret = NULL;
			exit(0xFA00);
		}

		ret->dataType.V.size = (*dimensions)[0];

		ret->dataType.V.vector = (float*)malloc((*dimensions)[0] * sizeof(float));
		RtlZeroMemory(ret->dataType.V.vector, (*dimensions)[0]);

		ret->dataType.V.objectReference = 1;		

		if (dataGenerateType == GENERATE_RANDOM)
		{
			fors(
				ret->dataType.V.size,
				ret->dataType.V.vector[j] = (float)(rand() % 20 + 1);
			);
		}
		else if (dataGenerateType == GENERATE_ALL_ZERO)
		{
			fors(
				ret->dataType.V.size,
				ret->dataType.V.vector[j] = 0.0;
			);
		}
		else if (dataGenerateType == GENERATE_ALL_ONE)
		{
			fors(
				ret->dataType.V.size,
				ret->dataType.V.vector[j] = 1.0;
			);
		}
		else if (dataGenerateType == GENERATE_TEST)
		{
			fors(
				ret->dataType.V.size,
				ret->dataType.V.vector[j] = (float)j;
			);
		}
		else if (dataGenerateType == GENERATE_EXISTING)
		{
			if (existData == NULL)
			{
				printf("ѡ�����Ѵ������ݽ��и��Ƶ�������ָ��Ϊ�գ��Ѳ���.\n");
				free(ret->dataType.V.vector);
				ret->dataType.V.vector = NULL;
				free(ret);
				ret = NULL;
				exit(0xFAC0);
			}
			else
			{
				fors(
					ret->dataType.V.size,
					ret->dataType.V.vector[j] = ((float*)existData)[j];
				);
			}
		}
		else
		{
			printf("�������Ͳ������Ѳ���.\n");
			free(ret->dataType.V.vector);
			ret->dataType.V.vector = NULL;
			free(ret);
			ret = NULL;
			exit(0xFAF0);
		}
	}
	else if (ret->dataFlag == CUDA_DATA_FLAG_$_MATRIX_$)
	{
		if ((*dimensions)[2] != 0)
		{
			printf("����ά�����������Ͳ������Ѳ���.\n");
			free(ret);
			ret = NULL;
			exit(0xFA00);
		}

		ret->dataType.M.rowSize = (*dimensions)[0];
		ret->dataType.M.colSize = (*dimensions)[1];

		ret->dataType.M.matrix = (float**)malloc((*dimensions)[0] * sizeof(float*));

		fors(
			(*dimensions)[0],
			ret->dataType.M.matrix[j] = (float*)malloc((*dimensions)[1] * sizeof(float));
		);
		forss(
			(*dimensions)[0], (*dimensions)[1],
			ret->dataType.M.matrix[j][i] = 0.0;,
		);

		ret->dataType.M.objectReference = 1;

		if (dataGenerateType == GENERATE_RANDOM)
		{
			forss(
				(*dimensions)[0], (*dimensions)[1],
				ret->dataType.M.matrix[j][i] = (float)(rand() % 20 + 1);,
			);
		}
		else if (dataGenerateType == GENERATE_ALL_ZERO)
		{
			forss(
				(*dimensions)[0], (*dimensions)[1],
				ret->dataType.M.matrix[j][i] = 0.0;,
			);
		}
		else if (dataGenerateType == GENERATE_ALL_ONE)
		{
			forss(
				(*dimensions)[0], (*dimensions)[1],
				ret->dataType.M.matrix[j][i] = 1.0;,
			);
		}
		else if (dataGenerateType == GENERATE_TEST)
		{
			forss(
				(*dimensions)[0], (*dimensions)[1],
				ret->dataType.M.matrix[j][i] = (float)(j * i);,
			);
		}
		else if (dataGenerateType == GENERATE_EXISTING)
		{
			if (existData == NULL)
			{
				printf("ѡ�����Ѵ������ݽ��и��Ƶ�������ָ��Ϊ�գ��Ѳ���.\n");
				
				fors(
					(*dimensions)[0],
					free(ret->dataType.M.matrix[j]);
					ret->dataType.M.matrix[j] = NULL;
				);
				free(ret->dataType.M.matrix);
				ret->dataType.M.matrix = NULL;
				exit(0xFAC0);
			}
			else
			{
				forss(
					(*dimensions)[0], (*dimensions)[1],
					ret->dataType.M.matrix[j][i] = ((float**)existData)[j][i];,
				);
			}
		}
		else
		{
			printf("�������Ͳ������Ѳ���.\n");
			fors(
				(*dimensions)[0],
				free(ret->dataType.M.matrix[j]);
				ret->dataType.M.matrix[j] = NULL;
			);
			free(ret->dataType.M.matrix);
			ret->dataType.M.matrix = NULL;
			exit(0xFAF0);
		}
	}
	else
	{
		printf("�������Ͳ������Ѳ���.\n");
		free(ret);
		ret = NULL;
		exit(0xC000);
	}
	return ret;
}

void checkDataSet(
	IN PDATA_SET dataSet
)
{
	if (dataSet->dataFlag == 1)
	{
		//vector
		if (!dataSet->dataType.V.objectReference)
		{
			printf("�������ü���Ϊ�㣬����Ϊ�գ��Ѳ���.\n");
		}
		printf("���ݼ�: ����\n");
		printf("��������: %zu\n", dataSet->dataType.V.size);
		printf("��ǰ���ü���: %zu\n", dataSet->dataType.V.objectReference);
		printf("������ֵ: \n");
		fors(
			dataSet->dataType.V.size,
			printf("%.3f\t", dataSet->dataType.V.vector[j]);
		);
		QAQ;
		return;
	}
	else if (dataSet->dataFlag == 2)
	{
		//matrix
		if (!dataSet->dataType.M.objectReference)
		{
			printf("�������ü���Ϊ�㣬����Ϊ�գ��Ѳ���.\n");
		}
		printf("���ݼ�: ����\n");
		printf("��������: %zu\n", dataSet->dataType.M.rowSize);
		printf("��������: %zu\n", dataSet->dataType.M.colSize);
		printf("��ǰ���ü���: %zu\n", dataSet->dataType.M.objectReference);
		printf("������ֵ: \n");
		forss(
			dataSet->dataType.M.rowSize, dataSet->dataType.M.colSize,
			printf("%.3f\t", dataSet->dataType.M.matrix[j][i]);,
			QAQ;
		);
	}
	else
	{
		printf("�Ƿ��������ͣ��Ѳ���.\n");
	}
	return;
}

void makeCudaFriendlyData(
	IN PDATA_SET hostDataSet,
	OUT_HOST_PTR float** deviceDataSet
)
{
	if (hostDataSet->dataFlag == CUDA_DATA_FLAG_$_VECTOR_$)
	{
		//vector
		if (!hostDataSet->dataType.V.objectReference)
		{
			printf("�������ü���Ϊ�㣬����Ϊ�գ��Ѳ���.\n");
		}
		*deviceDataSet = (float*)malloc(hostDataSet->dataType.V.size * sizeof(float));
		memcpy(*deviceDataSet, hostDataSet->dataType.V.vector, hostDataSet->dataType.V.size * sizeof(float));
		return;
	}
	else if (hostDataSet->dataFlag == CUDA_DATA_FLAG_$_MATRIX_$)
	{
		//matrix
		if (!hostDataSet->dataType.M.objectReference)
		{
			printf("�������ü���Ϊ�㣬����Ϊ�գ��Ѳ���.\n");
		}
		*deviceDataSet = (float*)malloc(hostDataSet->dataType.M.rowSize * hostDataSet->dataType.M.colSize * sizeof(float));
		for (size_t j = 0; j < hostDataSet->dataType.M.rowSize; j++)
		{
			for (size_t i = 0; i < hostDataSet->dataType.M.colSize; i++)
			{
				(*deviceDataSet)[i + j * hostDataSet->dataType.M.colSize] = hostDataSet->dataType.M.matrix[j][i];
			}
		}
	}
	else
	{
		printf("�Ƿ��������ͣ��Ѳ���.\n");
	}
	return;
}

void safetyCheck(
	IN PDATA_SET dataSet1,
	IN PDATA_SET dataSet2,
	IN myCudaDataFlag dataFlag,
	IN myCudaCalculateFlag calculateFlag
)
{
	if (dataSet1->dataFlag != dataFlag || dataSet2->dataFlag != dataFlag)
	{
		printf("�������ݼ������ͺ͵��õ����ݼ����Ͳ�ͬ���Ѳ���.\n");
		exit(0xECC0);
	}
	if (dataFlag == CUDA_DATA_FLAG_$_VECTOR_$)
	{
		if (calculateFlag == CUDA_CAU_TYPE_$_ADD_$)
		{
			if (dataSet1->dataType.V.size != dataSet2->dataType.V.size)
			{
				printf("���������ĳߴ粻ͬ���Ѳ���.\n");
				exit(0xDCC0);
			}
		}
		else if (calculateFlag == CUDA_CAU_TYPE_$_MULTIPLE_$)
		{
			if (dataSet1->dataType.V.size != dataSet2->dataType.V.size)
			{
				printf("���������ĳߴ粻ͬ���Ѳ���.\n");
				exit(0xDCC0);
			}
		}
		else
		{
			printf("���õļ������Ͳ���֧�֣��Ѳ���.\n");
			exit(0xDCC0);
		}
	}
	else if (dataFlag == CUDA_DATA_FLAG_$_MATRIX_$)
	{
		if (calculateFlag == CUDA_CAU_TYPE_$_ADD_$)
		{
			if
			(
				(dataSet1->dataType.M.rowSize != dataSet2->dataType.M.rowSize)
				||
				(dataSet2->dataType.M.colSize != dataSet2->dataType.M.colSize)
			)
			{
				printf("��ͼִ�о���ӷ�������������������в�����ͬ���Ѳ���.\n");
				exit(0xDCC8);
			}
		}
		else if (calculateFlag == CUDA_CAU_TYPE_$_MULTIPLE_$)
		{
			if (dataSet1->dataType.M.colSize != dataSet2->dataType.M.rowSize)
			{
				printf("��ͼִ�о���˷������ǵ�һ��������к͵ڶ���������в�����ͬ���Ѳ���.\n");
				exit(0xDCC8);
			}
		}
		else
		{
			printf("���õļ������Ͳ���֧�֣��Ѳ���.\n");
			exit(0xDCC0);
		}
	}
	else
	{
		printf("������������ʹ����Ѳ���.\n");
		exit(0xACC0);
	}
}

void cudaExFreeMem(
	IN_OUT PVOID* _GPU_mem
)
{
	hipFree(*_GPU_mem);
	*_GPU_mem = NULL;
	return;
}

void ExFreeMem(
	IN_OUT PVOID* mem
)
{
	free(*mem);
	*mem = NULL;
	return;
}

static void ExFreeDataSet_$_VECTOR_$(
	IN_OUT PDATA_SET* dataSet
)
{
	if ((*dataSet)->dataFlag != CUDA_DATA_FLAG_$_VECTOR_$)
	{
		printf("�������Ͳ������Ѳ���.\n");
		exit(0xCFF0);
	}
	ExFreeMem((PVOID*)&((*dataSet)->dataType.V.vector));
	return;
}

static void ExFreeDataSet_$_MATRIX_$(
	IN_OUT PDATA_SET* dataSet
)
{
	if ((*dataSet)->dataFlag != CUDA_DATA_FLAG_$_MATRIX_$)
	{
		printf("�������Ͳ������Ѳ���.\n");
		exit(0xCFF0);
	}
	fors(
		(*dataSet)->dataType.M.rowSize,
		ExFreeMem((PVOID*)&((*dataSet)->dataType.M.matrix[j]));
	);
	ExFreeMem((PVOID*)&((*dataSet)->dataType.M.matrix));
	return;
}

void ExFreeDataSet(
	IN_OUT PDATA_SET* dataSet
)
{
	switch ((*dataSet)->dataFlag)
	{
	case CUDA_DATA_FLAG_$_VECTOR_$:
		ExFreeDataSet_$_VECTOR_$(dataSet);
		break;
	case CUDA_DATA_FLAG_$_MATRIX_$:
		ExFreeDataSet_$_MATRIX_$(dataSet);
		break;
	default:
		printf("�Ƿ��������ͣ��Ѳ���.\n");
		break;
	}
	return;
}

void callCuda(
	IN PDATA_SET dataSet1,
	IN PDATA_SET dataSet2,
	IN myCudaDataFlag dataFlag,
	IN myCudaCalculateFlag calculateFlag,
	OUT_HOST_PTR float** ret
)
{
	safetyCheck(dataSet1, dataSet2, dataFlag, calculateFlag);

	float* _host_linerData1 = NULL;
	float* _host_linerData2 = NULL;

	makeCudaFriendlyData(dataSet1, &_host_linerData1);
	makeCudaFriendlyData(dataSet2, &_host_linerData2);

	float* _device_linerData1 = NULL;
	float* _device_linerData2 = NULL;

	float* _device_result = NULL;
	size_t _public_resultSize = 0x0;

	if (dataFlag == CUDA_DATA_FLAG_$_VECTOR_$ && calculateFlag == CUDA_CAU_TYPE_$_ADD_$)
	{
		_public_resultSize = dataSet1->dataType.V.size;
		hipMalloc(&_device_result, _public_resultSize * sizeof(float));
	}
	if (dataFlag == CUDA_DATA_FLAG_$_VECTOR_$ && calculateFlag == CUDA_CAU_TYPE_$_MULTIPLE_$)
	{
		_public_resultSize = 1;
		hipMalloc(&_device_result, _public_resultSize * sizeof(float));
	}
	if (dataFlag == CUDA_DATA_FLAG_$_MATRIX_$ && calculateFlag == CUDA_CAU_TYPE_$_ADD_$)
	{
		_public_resultSize = dataSet1->dataType.M.rowSize * dataSet1->dataType.M.colSize;
		hipMalloc(&_device_result, _public_resultSize * sizeof(float));
	}
	if (dataFlag == CUDA_DATA_FLAG_$_MATRIX_$ && calculateFlag == CUDA_CAU_TYPE_$_MULTIPLE_$)
	{
		_public_resultSize = dataSet1->dataType.M.rowSize * dataSet2->dataType.M.colSize;
		printf("_public_resultSize: %zu\n", _public_resultSize);
		hipMalloc(&_device_result, _public_resultSize * sizeof(float));

		hipMalloc(&_device_linerData1, dataSet1->dataType.M.rowSize * dataSet1->dataType.M.colSize * sizeof(float));
		hipMalloc(&_device_linerData2, dataSet2->dataType.M.rowSize * dataSet2->dataType.M.colSize * sizeof(float));

		hipMemcpy(_device_linerData1, _host_linerData1, dataSet1->dataType.M.rowSize * dataSet1->dataType.M.colSize * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(_device_linerData2, _host_linerData2, dataSet2->dataType.M.rowSize * dataSet2->dataType.M.colSize * sizeof(float), hipMemcpyHostToDevice);
		
		dim3 threadEx = { 0 };
		threadEx.x = 32;
		threadEx.y = 32;
		threadEx.z = 1;
		
		dim3 blockEx = { 0 };
		blockEx.x = 32;
		blockEx.y = 32;
		blockEx.z = 1;
		
		size_t data1_row = dataSet1->dataType.M.rowSize;
		size_t data1_col = dataSet1->dataType.M.colSize;
		size_t data2_row = dataSet2->dataType.M.rowSize;
		size_t data2_col = dataSet2->dataType.M.colSize;
		
		myCudaMatrixMultiple << <blockEx, threadEx >> > (_device_linerData1, _device_linerData2, data1_row, data1_col, data2_row, data2_col, _device_result);
	}
	//��������CUDA��ʽ��������Ϊ������ֵ������С��ǰ�ĵ�resultSize��ͬ��
	float* _host_result = NULL;
	_host_result = (float*)malloc(_public_resultSize * sizeof(float));
	if (_host_result != NULL)
	{
		hipMemcpy(_host_result, _device_result, _public_resultSize * sizeof(float), hipMemcpyDeviceToHost);
		*ret = _host_result;

		cudaExFreeMem((PVOID*)&_device_result);
		cudaExFreeMem((PVOID*)&_device_linerData2);
		cudaExFreeMem((PVOID*)&_device_linerData1);
		ExFreeMem((PVOID*)&_host_linerData2);
		ExFreeMem((PVOID*)&_host_linerData1);
	}
	else
	{		
		ExFreeMem((PVOID*)&_host_result);

		cudaExFreeMem((PVOID*)&_device_result);
		cudaExFreeMem((PVOID*)&_device_linerData2);
		cudaExFreeMem((PVOID*)&_device_linerData1);
		ExFreeMem((PVOID*)&_host_linerData2);
		ExFreeMem((PVOID*)&_host_linerData1);
		printf("�豸GPU�ڴ�������ָ��Ϊ�գ��Ѳ���.\n");
		*ret = NULL;
		exit(0xFFFF);
	}
	hipDeviceReset();
	return ;
}