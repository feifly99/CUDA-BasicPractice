#include "baseHeader.h"

int main()
{
	SIZE_T dimension1[MAX_DIMENSION_SIZE] = { 20480, 10240 };
	PDATA_SET dataSet1 = makeDataSet(CUDA_DATA_FLAG_$_MATRIX_$, GENERATE_ALL_ONE, &dimension1, NULL);
	SIZE_T dimension2[MAX_DIMENSION_SIZE] = { 10240, 20480 };
	PDATA_SET dataSet2 = makeDataSet(CUDA_DATA_FLAG_$_MATRIX_$, GENERATE_ALL_ONE, &dimension2, NULL);
	float* result = NULL;
	callCuda(dataSet1, dataSet2, CUDA_DATA_FLAG_$_MATRIX_$, CUDA_CAU_TYPE_$_MULTIPLE_$, &result);
	ExFreeMem((PVOID*)&result);
	ExFreeDataSet(&dataSet2);
	ExFreeDataSet(&dataSet1);
	return 0;
}